﻿#pragma once
#include "HawkCams.cuh"
#include <crow.h>
#include <unordered_set>
#include <iostream>
#include <thread>
#include <chrono>

void working_code(std::unordered_set<crow::websocket::connection*>& clients) {

    const short int camsCount = 3;
    std::string paths[camsCount] = { "videos\\v1.mp4", "videos\\v2.mp4", "videos\\v3.mp4"};
    HawkCam cams(paths, camsCount);
    cams.loadStreams();
    cams.setupStreams();
    cams.loop(clients);
    cams.finalize();
}

int main() {
    crow::SimpleApp app;

    // Define a set to keep track of connected clients
    std::unordered_set<crow::websocket::connection*> clients;

    std::thread broadcastThread(working_code, std::ref(clients));

    CROW_ROUTE(app, "/image_frame")
        .websocket()
        .onopen([&clients](crow::websocket::connection& conn) {
        clients.insert(&conn);
            std::cout << "New connection established" << std::endl;
        })
        .onclose([&clients](crow::websocket::connection& conn, const std::string& /*reason*/) {
            clients.erase(&conn);
            std::cout << "Connection closed" << std::endl;
        });

        // Start the server on port 8080
        app.port(8080).multithreaded().run();

        // Wait for the broadcast thread to finish (which will never happen)
        broadcastThread.join();

    return 0;
}
