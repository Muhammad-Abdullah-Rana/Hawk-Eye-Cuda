﻿#pragma once
#include "Segmentation.h"
#include "ObjectIdentification.cuh"

void cudaCall(VideoCapture& vid) {
    Mat img;
    hipError_t cudaStatus;
    vid.read(img);
    while (!img.empty()) {
        cudaStatus = cudaFitTransform(img, 30);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "cuda failed!");
            return;
        }
        imshow("vid", img);
        waitKey(5);
        vid.read(img);
    }
}

int main()
{
    VideoCapture vid("vid001.y4m");
    cudaCall(vid);
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
