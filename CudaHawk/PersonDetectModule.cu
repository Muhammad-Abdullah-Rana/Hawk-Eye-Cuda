#include "hip/hip_runtime.h"
#include "PersonDetectModule.cuh"

PersonDetect::PersonDetect(){
	camId = -1;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}
PersonDetect::PersonDetect(int cid) {
	camId = cid;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}
bool PersonDetect::shouldEnd() { return cudaStatus != hipSuccess; }
bool PersonDetect::fit(Mat& curr) {
	_count++;
	frame = curr.clone();
	Mat grayFrame;
	basics.convertGrey(curr, grayFrame);
	basics.checkDiffrence(grayFrame, bgModel, fgMask);
	bgModel = 0.9 * bgModel + 0.1 * grayFrame;
	basics.removeNoise(fgMask, 40);
	return true;
}

void print(Location l) {
	fprintf(stderr, "CamId: %d, PId: %d, time: %d, loc.x: %f,  loc.y %f \n",l.camId, l.pid, l.time, l.locX, l.locY);
}

void PersonDetect::save(const Mat& personImage, int id) {
	// Placeholder function to save person image and ID to a database
	// Implement the logic to save the image and ID to your specific database
	if (_show) {
		string str = "ID: ";
		str += (id);
		cv::imshow(str, personImage);
	}
}
void PersonDetect::log(Person p) {
	Location loc;
	loc.pid = p.id;
	loc.camId = camId;
	loc.locX = p.centroid.x;
	loc.locY = p.centroid.y;
	auto currentTime = system_clock::now().time_since_epoch();
	loc.time = duration_cast<seconds>(currentTime).count();
	print(loc);
}
void PersonDetect::showDetectedPerson(bool _s) {
	_show = _s;
}
bool PersonDetect::transform(){
	//imshow("mask", fgMask);
	// waiting for background learning
	if (_count < FRAME_THRESH_COUNT) {
		return false;
	}
	vector<std::vector<cv::Point>> contours;
	basics.findContours(fgMask, contours);
	
	// Process each contour to detect and track persons
	for (const auto& contour : contours) {
		double area = cv::contourArea(contour);
		cv::Rect boundingRect = cv::boundingRect(contour);
		//if (boundingRect.height / boundingRect.width >= 1.25f) 
		if (area > 500 ) {
			Person person = getPerson(frame, boundingRect);
			Mat personImage = frame(person.boundingRect).clone();
			save(personImage, person.id);
			log(person);

			// Draw bounding box with person ID on the frame
			cv::rectangle(frame, person.boundingRect, cv::Scalar(0, 255, 0), 2);
			cv::putText(frame, std::to_string(person.id), cv::Point(person.boundingRect.x, person.boundingRect.y - 10),
				cv::FONT_HERSHEY_SIMPLEX, 0.9, cv::Scalar(0, 255, 0), 2);;
		}
	}
	return !shouldEnd();
}
Mat PersonDetect::getOrignalFrame() { return frame; }

void PersonDetect::finilize(){
	if (shouldEnd())
		cout << hipGetErrorString(cudaStatus) << endl;

	cudaStatus = hipDeviceReset();

}

float PersonDetect::calculateCentroidDistance(const cv::Point2f& centroid1, const cv::Point2f& centroid2) {
	return cv::norm(centroid1 - centroid2);
}
Person PersonDetect::getPerson(Mat& frame, cv::Rect boundingRect) {
	cv::Point2f centroid = (boundingRect.tl() + boundingRect.br()) * 0.5f;
	

	for (auto& pair : persons) {
		Person& person = pair.second;

		// Calculate distance between centroids
		float distance = calculateCentroidDistance(person.centroid, centroid);

		// Check if the distance is smaller than a threshold
		if (distance < 50) {
			person.boundingRect = boundingRect;
			person.centroid = centroid;
			persons[personId] = person;
			return person;
		}
	}

	// If person not found, assign a new ID and add to the list
	Person newPerson;
	newPerson.id = personId;
	newPerson.boundingRect = boundingRect;
	newPerson.centroid = centroid;
	persons[personId] = newPerson;

	personId++;
	return newPerson;
}
